#include "hip/hip_runtime.h"
// Adapted from Section 8.2 of "Cuda by Example" 2010 
// Compile it as standalone with: 
// /Developer/NVIDIA/CUDA-6.0/bin/nvcc -arch=sm_30 -o ogltest ogltest.cu -lglut
// 

#if __unix__
#define GL_GLEXT_PROTOTYPES 1
#include <GL/gl.h>
#include <GL/glext.h>
#endif

#include <stdio.h> 
#include <stdlib.h> 
#include <string.h> 

#if __unix__
#include <GL/gl.h> 
#include <GL/glut.h>
#else
#include <OpenGL/gl.h> 
#include <OpenGL/glut.h>
#endif

#include <cuda_gl_interop.h> 
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

//function to check if a CUDA error has been raised
#define cutool_checkErrorMsg()   __cutool_checkErrorMsg( __FILE__, __LINE__)
inline void __cutool_checkErrorMsg( const char *file, const int line )
{
	hipError_t cudares = hipGetLastError();
    if( hipSuccess != cudares)
	{
		fprintf(stderr, "CUDA Runtime API error (file %s, line %i): %s.\n",file, line, hipGetErrorString( cudares) );
        exit(-1);
    }
}

#define GET_PROC_ADDRESS( str ) glXGetProcAddress( (const GLubyte *)str ) 

static void HandleError( hipError_t err, const char *file,  int line ) { 
    if (err != hipSuccess) { 
            printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line ); 
            exit( EXIT_FAILURE ); 
    } 
} 
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 

#define     DIM    512 

GLuint  bufferObj; 
hipGraphicsResource *resource; 

struct sort_functor
{
  __host__ __device__
    bool operator()(uchar4 left, uchar4 right) const
    {
      return (left.y < right.y);
    }
};


// create a green/black pattern
__global__ void kernel( uchar4 *ptr ) { 
// map from threadIdx/BlockIdx to pixel position 
  int x = threadIdx.x + blockIdx.x * blockDim.x; 
  int y = threadIdx.y + blockIdx.y * blockDim.y; 
  int offset = x + y * blockDim.x * gridDim.x; 

// now calculate the value at that position 
  float fx = x/(float)DIM - 0.5f; 
  float fy = y/(float)DIM - 0.5f; 
  unsigned char   green = 128 + 127 * sin( abs(fx*100) - abs(fy*100) ); 

// accessing uchar4 vs unsigned char* 
  ptr[offset].x = 0; 
  ptr[offset].y = green; 
  ptr[offset].z = 0; 
  ptr[offset].w = 255; 
} 


#ifdef CPU_DATA
void fill_data( uchar4 *ptr ) {
    for(int x=0; x<DIM; x++){
        for(int y=0; y<DIM; y++){
            int offset = x + y * DIM;
            float fx = x/(float)DIM - 0.5f; 
            float fy = y/(float)DIM - 0.5f; 
            unsigned char   green = 128 + 127 * sin( abs(fx*100) - abs(fy*100) ); 
          
          // accessing uchar4 vs unsigned char* 
            ptr[offset].x = 0; 
            ptr[offset].y = green; 
            ptr[offset].z = 0; 
            ptr[offset].w = 255;           
        }
    } 
} 
#endif


static void draw_func( void ){
#ifndef CHECK_OPENGL_WORKS 
  glDrawPixels( DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0 );
#else
    glClear(GL_COLOR_BUFFER_BIT);
    glClearColor(0,0,0,1);
    float x=-.5, y=-.5;
    float w=1, h=1;    
    glColor4f(1, 1, 1, 1);
    glBegin(GL_QUADS);
        glVertex3f(x,y,0);
        glVertex3f(x+w,y,0);
        glVertex3f(x+w,y+h,0);
        glVertex3f(x,y+h,0);
    glEnd();
#endif
    
  glutSwapBuffers(); 
}

static void sort_pixels(){
    hipGraphicsMapResources( 1, &resource, NULL ); 
        uchar4* devPtr; 
        size_t  size; 
        hipGraphicsResourceGetMappedPointer( (void**)&devPtr, &size, resource); 
        thrust::device_ptr<uchar4> tptr = thrust::device_pointer_cast(devPtr);
        thrust::sort(tptr, tptr+(DIM*DIM), sort_functor());
    hipGraphicsUnmapResources( 1, &resource, NULL ); 
    draw_func();
}

static void key_func( unsigned char key, int x, int y ) { 
  switch (key) { 
    case 27: 
        HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) ); 
        glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 ); 
        glDeleteBuffers( 1, &bufferObj ); 
        exit(0); 
        break;
    case 32: /// spacebar
        sort_pixels();
        break;
    default:
        break;
  } 
} 



int main(int argc, char *argv[]) { 
    cudaGLSetGLDevice( 0 ); 
    glutInit( &argc, argv ); 
    glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA ); 
    glutInitWindowSize( DIM, DIM ); 
    glutCreateWindow( "cuda_thrust_sort" );

#ifdef CPU_DATA
    /// Do the same operation on CPU instead of GPU
    uchar4 data[DIM*DIM];
    fill_data(data);
#else
    uchar4* data = NULL;
#endif
  
    ///--- Generate an OpenGL pixel buffer to write stuff on
    glGenBuffers( 1, &bufferObj ); 
    glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj ); 
    glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4, data, GL_DYNAMIC_DRAW_ARB ); 
    
    ///--- Registers it with cuda (devPtr) so that we can process it  
    hipGraphicsGLRegisterBuffer( &resource, bufferObj, cudaGraphicsMapFlagsNone ); 
    hipGraphicsMapResources( 1, &resource, NULL ); 
    uchar4* devPtr; ///< cuda accessed pointer
    size_t  size; 
    hipGraphicsResourceGetMappedPointer( (void**)&devPtr, &size, resource); 
    
    ///--- Do something with it 
    dim3    grid(DIM/16,DIM/16); 
    dim3    threads(16,16); 
    kernel<<<grid,threads>>>( devPtr ); 
    cutool_checkErrorMsg();
    hipGraphicsUnmapResources( 1, &resource, NULL ); 
    
    ///--- Display result using OpenGL directly
    glutKeyboardFunc( key_func ); 
    glutDisplayFunc( draw_func ); 
    glutMainLoop(); 
} 
