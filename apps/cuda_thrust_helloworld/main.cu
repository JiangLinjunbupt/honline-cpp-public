#include "hip/hip_runtime.h"
﻿#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <thrust/transform.h> 
#include <thrust/replace.h> 
#include <thrust/functional.h>
#include <thrust/count.h>
#include <thrust/scan.h>

#include <iostream>
#include <fstream>

#include <thrust/iterator/transform_iterator.h>
#include <thrust/adjacent_difference.h>

#include <queue>

/*
VECTORS
Thrust provides two containters for vector:
- host_vector - stored in CPU
- device_vector - stored in GPU
otherwise, they are exactly like std::vector.
*/
void thrust_vectors_introduction() {

	// INRODUCTION
	thrust::host_vector<int> H(4);
	H[0] = 14; H[1] = 20; H[2] = 38; H[3] = 46;
	for (int i = 0; i < H.size(); i++)
		std::cout << "H[" << i << "] = " << H[i] << " ";
	std::cout << std::endl;

	// Operator "=" is used to copy_host vector to device_vector;
	thrust::device_vector<int> D = H;

	// Each access through operator "[]" requires use of hipMemcpy, thus use sparingly
	D[0] = 99; D[1] = 88;

	// print contents of D
	for (int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << " ";
}

void thrust_vectors_all_elements() {

	// initialize all ten integers of a device_vector to 1 
	thrust::device_vector<int> D(10, 1);
	// set the first seven elements of a vector to 9 
	thrust::fill(D.begin(), D.begin() + 7, 9);
	// initialize a host_vector with the first five elements of D 
	thrust::host_vector<int> H(D.begin(), D.begin() + 5);
	// set the elements of H to 0, 1, 2, 3, ... 
	thrust::sequence(H.begin(), H.end());
	// copy all of H back to the beginning of D 
	thrust::copy(H.begin(), H.end(), D.begin());
	//print D 
	for (int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

void iterators_and_static_dispatching() {
	/* You can pass not only iterators like H.begin() by also raw pointers
	to thrust:: functions. A regular pointer is considered by the library to be
	pointing to CPU location (if it is in fact pointing to GPU this will cause problems).
	To tell the library that the pointer actually points to a GPU location,
	wrap it with thrust::device_ptr.
	*/

	size_t N = 10;
	// raw pointer to device memory 
	int * raw_ptr;
	hipMalloc((void **)&raw_ptr, N * sizeof(int)); //allocates memory at GPU
	// wrap raw pointer with a device_ptr 
	thrust::device_ptr<int> dev_ptr(raw_ptr);
	// use device_ptr in thrust algorithms 
	thrust::fill(dev_ptr, dev_ptr + N, (int)0);

	// extracting the row pointer
	raw_ptr = thrust::raw_pointer_cast(dev_ptr);
}

// saxply is a BLAS function : y <- a * x + y
struct saxpy_functor {
	const float a;
	saxpy_functor(float _a) : a(_a) {}
	__host__ __device__
		float operator()(const float& x, const float& y) const {
		return a * x + y;
	}
};


void thrust_algorithms_transformations() {
	// allocate three device_vectors with 10 elements 
	thrust::device_vector<int> X(10);
	thrust::device_vector<int> Y(10);
	thrust::device_vector<int> Z(10);
	// initialize X to 0,1,2,3, .... 
	thrust::sequence(X.begin(), X.end());
	// compute Y = -X 
	thrust::transform(X.begin(), X.end(), Y.begin(), thrust::negate<int>());
	// fill Z with twos
	thrust::fill(Z.begin(), Z.end(), 2);
	// compute Y = X mod 2 
	thrust::transform(X.begin(), X.end(), Z.begin(), Y.begin(), thrust::modulus<int>());
	// replace all the ones in Y with tens 
	thrust::replace(Y.begin(), Y.end(), 1, 10);
	// print Y 
	thrust::copy(Y.begin(), Y.end(), std::ostream_iterator<int>(std::cout, "\n"));

	float A = 5;
	// Above the saw build-in transformations, now consider a user-defined transformation
	thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
	// thrust::transform only supports transformations with one or two input 
	// arguments(e.g.f(x) → y and f(x, x) → y).

}

void thrust_algorithms_reductions() {
	thrust::device_vector<int> D(10, 1);
	int sum = thrust::reduce(D.begin(), D.end(), (int)0 /*initial value*/, thrust::plus<int>());

	// count the 1s in a vector
	thrust::device_vector<int> vec(5, 0);
	vec[1] = 1; vec[3] = 1; vec[4] = 1;
	int result = thrust::count(vec.begin(), vec.end(), 1);

}

template <typename T> struct square {
	__host__ __device__
		T operator()(const T& x) const {
		return x * x;
	}
};

void thrust_algorithms_compute_norm_example() {
	// initialize host array 
	float x[4] = { 1.0, 2.0, 3.0, 4.0 };
	// transfer to device 
	thrust::device_vector<float> d_x(x, x + 4);
	float norm = std::sqrt(
		thrust::transform_reduce(d_x.begin(), d_x.end(), square<float>(), 0, thrust::plus<float>())
		);
	std::cout << norm << std::endl;
}

template<typename T>
struct absolute_value {
	__host__ __device__
		T operator()(const T &x) const {
		return x < T(0) ? -x : x;
	}
};

void sum_of_abs() {
	thrust::device_vector<int> X(10);
	thrust::sequence(X.begin(), X.end());
	thrust::transform(X.begin(), X.begin() + 5, X.begin(), thrust::negate<int>());
	int result = thrust::transform_reduce(X.begin(), X.end(), absolute_value<int>(), 0, thrust::plus<int>());
	thrust::copy(X.begin(), X.end(), std::ostream_iterator<int>(std::cout, " "));
	std::cout << "result = " << result << std::endl;
}


struct square_root : public thrust::unary_function < float, float > {
	__host__ __device__
		float operator()(float x) const {
		return sqrtf(x);
	}
};


struct even : public thrust::unary_function < int, int > {
	__host__ __device__
		int operator()(int x) {
		if (x % 2 == 0) return x;
		else return 1;
	}
};

struct root_of_sum_of_squares : public thrust::binary_function < float, float, float > {
	__host__ __device__
		float operator()(float x, float y) {
		return sqrt(x * x + y * x);
	}
};

struct erase_evens : public thrust::binary_function < float, float, int > {
	__host__ __device__
		float operator()(float x, int i) {
		if (i % 2 == 1)
			return x;
		else return 0;
	}
};



void norm_of_unrolled_vector() {


	int h_data[8] = { 1, 2, 1, 2, 1, 2, 1, 2 };
	thrust::device_vector<int> d_data(h_data, h_data + 8);
	thrust::device_vector<float> d_result(8);
	thrust::device_vector<float> final(8);

	thrust::adjacent_difference(d_data.begin(), d_data.end(), d_result.begin(), root_of_sum_of_squares());
	thrust::counting_iterator<int> count(8);
	thrust::transform(d_result.begin(), d_result.end(), count, d_result.begin(), erase_evens());
	//thrust::copy(final.begin(), final.end(), std::ostream_iterator<float>(std::cout, "\n"));
	std::cout << thrust::reduce(d_result.begin(), d_result.end()) << std::endl;

}



int main(void) {
	thrust::device_vector<float> device_pointer;
	device_pointer.resize(10);
	thrust::fill(device_pointer.begin(), device_pointer.end(), 0);
	float * raw_pointer = thrust::raw_pointer_cast(device_pointer.data());
	std::cout << raw_pointer[0] << std::endl;
	//Pull functor_pull(raw_pointer);
	//thrust::device_vector<int> offsets;
	//offsets.resize(camera_width * camera_heigth);
	//thrust::sequence(offsets.begin(), offsets.end());
	//thrust::for_each(offsets.begin(), offsets.end(), functor_pull);

	//float * output = thrust::raw_pointer_cast(&device_pointer[0]);

	

	/*
	Full documentation
	http://thrust.github.io/doc/group__transformations.html
	Tutorial
	http://docs.nvidia.com/cuda/thrust/#axzz3WiiNhwRB
	*/

	/*thrust_vectors_introduction();
	thrust_vectors_all_elements();
	iterators_and_static_dispatching();
	thrust_algorithms_transformations();
	thrust_algorithms_reductions();
	thrust_algorithms_compute_norm_example();*/

	//norm_of_unrolled_vector();
	float pull_moving_sum = 0;
	int moving_window_size = 5;
	std::queue<float> pull_error_history;
	for (size_t i = 0; i < 10; i++) {
		float pull_error = i;
		pull_error_history.push(pull_error);
		std::cout << "i = " << i << ", size = " << pull_error_history.size() << std::endl;
		pull_moving_sum = pull_moving_sum + pull_error;
		if (pull_error_history.size() > moving_window_size) {
			pull_moving_sum = pull_moving_sum - pull_error_history.front();
			pull_error_history.pop();
		}
		float pull_moving_average = pull_moving_sum / pull_error_history.size();
		std::cout << pull_moving_average << std::endl;
	}



	return 0;
}
