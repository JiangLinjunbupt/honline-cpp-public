#include "util/gl_wrapper.h" ///< for cuda_gl_interop
#include <cuda_gl_interop.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/count.h>

#include "cudax/kernel.h"
#include "cudax/CudaTimer.h"
#include "cudax/hip/hip_runtime_api.h" ///< SDK error checking
#include "cudax/CublasHelper.h"
#include "cudax/CudaHelper.h"
#include "cudax/KinectCamera.h"
#include "cudax/kernel_init.h"
#include "cudax/kernel_upload.h"
#include "cudax/kernel_debug.h"

#include "cudax/functors/ComputeJacobianSilhouette.h"
#include "cudax/functors/ComputeJacobianData.h"

#include <iostream>
#include <fstream>
#include <string>

using namespace cudax;

struct absolute_value : public thrust::unary_function < float, float > {
	__host__ __device__
		float operator()(float x) const {
		return (x >= 0) ? x : -x;
	}
};

void kernel_bind() {
	if (cudax::sensor_depth_array)
		CHECK_CUDA(hipBindTextureToArray(sensor_depth_texture_cuda, cudax::sensor_depth_array));
	if (settings->fit2D_silhouette_enable) {
		if (cudax::rendered_block_indices_array)
			CHECK_CUDA(hipBindTextureToArray(rendered_block_indices_texture_cuda, cudax::rendered_block_indices_array));
		if (cudax::rendered_depth_array)
			CHECK_CUDA(hipBindTextureToArray(rendered_depth_texture_cuda, cudax::rendered_depth_array));
	}
}

void kernel_unbind() {
	if (cudax::sensor_depth_array) CHECK_CUDA(hipUnbindTexture(sensor_depth_texture_cuda));
	cudax::sensor_depth_array = NULL;
	if (settings->fit2D_silhouette_enable) {
		if (cudax::rendered_block_indices_array) CHECK_CUDA(hipUnbindTexture(rendered_block_indices_texture_cuda));
		cudax::rendered_block_indices_array = NULL;
		if (cudax::rendered_depth_array) CHECK_CUDA(hipUnbindTexture(rendered_depth_texture_cuda));
		cudax::rendered_depth_array = NULL;
	}
}

void multiply_on_cpu(float* eigen_JtJ, float* eigen_JtF, thrust::device_vector<float> * J, thrust::device_vector<float>* F, int n_total, size_t frame_id, size_t iter) {
	thrust::host_vector<float> F_host(n_total);
	thrust::copy(F->begin(), F->begin() + n_total, F_host.begin());
	thrust::host_vector<float> J_host(n_total * _num_parameters);
	thrust::copy(J->begin(), J->begin() + n_total * _num_parameters, J_host.begin());

	for (size_t i = 0; i < _num_parameters; i++) {
		double result = 0;
		for (size_t k = 0; k < n_total; k++) {
			result = result + (double)J_host[k * _num_parameters + i] * (double)F_host[k];
		}
		eigen_JtF[i] = result;
	}
	for (size_t i = 0; i < _num_parameters; i++) {
		for (size_t j = 0; j < _num_parameters; j++) {
			double result = 0;
			for (size_t k = 0; k < n_total; k++) {
				result = result + (double)J_host[k * _num_parameters + i] * (double)J_host[k * _num_parameters + j];
			}
			eigen_JtJ[i * _num_parameters + j] = result;
		}
	}
	
	if (settings->write_jacobian_to_file) {
		// write jacobian to file
		std::ofstream output_file;
		std::string data_path = "C:/Users/tkach/Desktop/Test/";
		output_file.open(data_path + "j-" + std::to_string(frame_id) + "-" + std::to_string(iter) + ".txt");
		for (size_t i = 0; i < n_total; i++) {
			for (size_t k = 0; k < _num_parameters; k++) {
				output_file << (double)J_host[k * _num_parameters + i] << " ";
			}
		}
		output_file.close();

		// write function to file
		output_file.open(data_path + "f-" + std::to_string(frame_id) + "-" + std::to_string(iter) + ".txt");
		for (size_t i = 0; i < n_total; i++) {
			output_file << (double)F_host[i] << std::endl;
		}
		output_file.close();
	}
	
}

float compute_weighted_metric(int num_sensor_points, int num_rendered_points) {

	//float alpha_fingers = 0.8; float alpha_palm = 1 - alpha_fingers;
	//float alpha_sensor = 0.8; float alpha_rendered = 1 - alpha_sensor;

	float alpha_fingers = 0.5; float alpha_palm = 1 - alpha_fingers;
	float alpha_sensor = 1; float alpha_rendered = 1 - alpha_sensor;

	/// Sensor metrics	
	thrust::transform(_weighted_metric_sensor_palm->begin(), _weighted_metric_sensor_palm->begin() + num_sensor_points, _weighted_metric_sensor_palm->begin(), absolute_value());
	float error_sensor_palm = thrust::reduce(_weighted_metric_sensor_palm->begin(), _weighted_metric_sensor_palm->begin() + num_sensor_points);
	int num_sensor_points_palm = num_sensor_points - thrust::count(_weighted_metric_sensor_palm->begin(), _weighted_metric_sensor_palm->begin() + num_sensor_points, 0);
		
	thrust::transform(_weighted_metric_sensor_fingers->begin(), _weighted_metric_sensor_fingers->begin() + num_sensor_points, _weighted_metric_sensor_fingers->begin(), absolute_value());
	float error_sensor_fingers = thrust::reduce(_weighted_metric_sensor_fingers->begin(), _weighted_metric_sensor_fingers->begin() + num_sensor_points);
	int num_sensor_points_fingers = num_sensor_points - thrust::count(_weighted_metric_sensor_fingers->begin(), _weighted_metric_sensor_fingers->begin() + num_sensor_points, 0);

	//std::cout << "num_sensor_points = " << num_sensor_points << ", num_sensor_points_palm = " << num_sensor_points_palm << ", num_sensor_points_fingers = " << num_sensor_points_fingers <<
	//	", sum = " << num_sensor_points_palm + num_sensor_points_fingers << std::endl;
	
	/// Rendered metrics	
	thrust::transform(_weighted_metric_rendered_palm->begin(), _weighted_metric_rendered_palm->begin() + num_rendered_points, _weighted_metric_rendered_palm->begin(), absolute_value());
	float error_rendered_palm = thrust::reduce(_weighted_metric_rendered_palm->begin(), _weighted_metric_rendered_palm->begin() + num_rendered_points);
	int num_rendered_points_palm = num_rendered_points - thrust::count(_weighted_metric_rendered_palm->begin(), _weighted_metric_rendered_palm->begin() + num_rendered_points, 0);

	thrust::transform(_weighted_metric_rendered_fingers->begin(), _weighted_metric_rendered_fingers->begin() + num_rendered_points, _weighted_metric_rendered_fingers->begin(), absolute_value());
	float error_rendered_fingers = thrust::reduce(_weighted_metric_rendered_fingers->begin(), _weighted_metric_rendered_fingers->begin() + num_rendered_points);
	int num_rendered_points_fingers = num_rendered_points - thrust::count(_weighted_metric_rendered_fingers->begin(), _weighted_metric_rendered_fingers->begin() + num_rendered_points, 0);

	//std::cout << "num_rendered_points = " << num_rendered_points << ", num_rendered_points_palm = " << num_rendered_points_palm << ", num_rendered_points_fingers = " << num_rendered_points_fingers <<
	//	", sum = " << num_rendered_points_palm + num_rendered_points_fingers << std::endl << std::endl;

	/// Weighted metric
	float weighted_error_sensor = (alpha_palm * error_sensor_palm + alpha_fingers * error_sensor_fingers) / (alpha_palm * num_sensor_points_palm + alpha_fingers * num_sensor_points_fingers);
	float weighted_error_rendered = (alpha_palm * error_rendered_palm + alpha_fingers * error_rendered_fingers) / (alpha_palm * num_rendered_points_palm + alpha_fingers * num_rendered_points_fingers);

	float weighted_error = alpha_sensor * weighted_error_sensor + alpha_rendered * weighted_error_rendered;

	return weighted_error;
}

void kernel(float* eigen_JtJ, float* eigen_JtF, float & push_error, float & pull_error, float & weighted_error, bool eval_metric,
	bool reweight, int frame_id, int iter, int num_sensor_points, int num_rendered_points,
	float * correspondences_data_points, float * correspondences_model_points, int * correspondences_model_indices, int * correspondences_block_indices, float * jacobian_data,
	float * correspondences_silhouette_data_points, float * correspondences_silhouette_model_points, int * correspondences_silhouette_indices, float * jacobian_silhouette) {

	int n_pull, n_push;

	if (settings->fit2D_outline_enable || settings->fit2D_silhouette2outline_enable || settings->fit2D_silhouette_enable) {
		if (settings->fit2D_unproject) {
			n_push = 1 * num_rendered_points;
		}
		else {
			n_push = 2 * num_rendered_points;
		}
		n_pull = 1 * num_sensor_points;
	}
	else {
		num_rendered_points = 0;
		n_push = 0;
		n_pull = 1 * num_sensor_points; 
	}

	int n_total = n_pull + n_push;
	thrust::fill(J->begin(), J->begin() + n_total * _num_parameters, 0.0f);
	thrust::fill(F->begin(), F->begin() + n_total, 0.0f);
	if (_compute_weighted_metric) {
		thrust::fill(_weighted_metric_sensor_palm->begin(), _weighted_metric_sensor_palm->begin() + num_sensor_points, 0.0f);
		thrust::fill(_weighted_metric_sensor_fingers->begin(), _weighted_metric_sensor_fingers->begin() + num_sensor_points, 0.0f);
		thrust::fill(_weighted_metric_rendered_palm->begin(), _weighted_metric_rendered_palm->begin() + num_rendered_points, 0.0f);
		thrust::fill(_weighted_metric_rendered_fingers->begin(), _weighted_metric_rendered_fingers->begin() + num_rendered_points, 0.0f);
	}
	if (n_total == 0) return;

	float * J_push = thrust::raw_pointer_cast(J->data());
	float * J_pull = J_push + n_push * _num_parameters;
	float* F_push = thrust::raw_pointer_cast(F->data());
	float* F_pull = F_push + n_push;

	/// Parallel run
	ComputeJacobianSilhouette jacobian_silhouette_functor(J_push, F_push, _verify_jacobian, _compute_weighted_metric);
	ComputeJacobianData jacobian_data_functor(J_pull, F_pull, reweight, _verify_jacobian, _compute_weighted_metric);
	
	if (settings->fit2D_outline_enable || settings->fit2D_silhouette2outline_enable || settings->fit2D_silhouette_enable) {
		thrust::sequence(push_indices->begin(), push_indices->begin() + num_rendered_points);
		thrust::for_each(push_indices->begin(), push_indices->begin() + num_rendered_points, jacobian_silhouette_functor);
	}
	if (settings->fit3D_enable) {
		thrust::sequence(pull_indices->begin(), pull_indices->begin() + num_sensor_points);
		thrust::for_each(pull_indices->begin(), pull_indices->begin() + num_sensor_points, jacobian_data_functor);
	}
	
	/// Multiply with GPU
	CublasHelper::outer_product_J(*J, *JtJ, n_total, _num_parameters);
	thrust::copy(JtJ->begin(), JtJ->end(), eigen_JtJ);
	CublasHelper::vector_product_J(*J, *F, *JtF, n_total, _num_parameters);
	thrust::copy(JtF->begin(), JtF->end(), eigen_JtF);

	/// Multiply with CPU
	if (_multiply_on_cpu) {
		multiply_on_cpu(eigen_JtJ, eigen_JtF, J, F, n_total, frame_id, iter);
	}

	/// Only need evaluate metric on the last iteration
	if (eval_metric) {
		thrust::device_vector<float> f_pull(n_pull);
		thrust::transform(F->begin() + n_push, F->begin() + n_push + n_pull, f_pull.begin(), absolute_value());
		pull_error = thrust::reduce(f_pull.begin(), f_pull.end());
		pull_error = pull_error / n_pull;
		thrust::device_vector<float> f_push(n_push);
		thrust::transform(F->begin(), F->begin() + n_push, f_push.begin(), absolute_value());
		push_error = thrust::reduce(f_push.begin(), f_push.end());
		push_error = push_error / n_push;

		if (_compute_weighted_metric) weighted_error = compute_weighted_metric(num_sensor_points, num_rendered_points);
	}

	//Return the correspondences	
	if (_verify_jacobian) {
		thrust::copy(_correspondences_data_points->begin(), _correspondences_data_points->begin() + num_sensor_points * 3, correspondences_data_points);
		thrust::copy(_correspondences_model_points->begin(), _correspondences_model_points->begin() + num_sensor_points * 3, correspondences_model_points);
		thrust::copy(_correspondences_model_indices->begin(), _correspondences_model_indices->begin() + num_sensor_points * 3, correspondences_model_indices);
		thrust::copy(_correspondences_block_indices->begin(), _correspondences_block_indices->begin() + num_sensor_points, correspondences_block_indices);
		thrust::copy(_jacobian_data->begin(), _jacobian_data->begin() + num_sensor_points * 3 * _num_parameters, jacobian_data);

		thrust::copy(_correspondences_silhouette_data_points->begin(), _correspondences_silhouette_data_points->begin() + num_rendered_points * 3, correspondences_silhouette_data_points);
		thrust::copy(_correspondences_silhouette_model_points->begin(), _correspondences_silhouette_model_points->begin() + num_rendered_points * 3, correspondences_silhouette_model_points);
		thrust::copy(_correspondences_silhouette_indices->begin(), _correspondences_silhouette_indices->begin() + num_rendered_points * 3, correspondences_silhouette_indices);
		thrust::copy(_jacobian_silhouette->begin(), _jacobian_silhouette->begin() + num_rendered_points * 3 * _num_parameters, jacobian_silhouette);
	}
}
